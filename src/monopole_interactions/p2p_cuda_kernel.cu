
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_CUDA_ENABLED
#include <sstream>
#include "monopole_kernel_templates.hpp"
#include "p2p_cuda_kernel.hpp"
namespace octotiger {
namespace fmm {
    namespace monopole_interactions {
        // __constant__ octotiger::fmm::multiindex<> device_stencil_const[P2P_PADDED_STENCIL_SIZE];
        __constant__ double device_stencil_masks[FULL_STENCIL_SIZE];
        __constant__ double device_four_constants[FULL_STENCIL_SIZE * 4];

        void copy_stencil_to_p2p_constant_memory(const double *stencil_masks, const size_t full_stencil_size) {
            cudaError_t err = cudaMemcpyToSymbol(device_stencil_masks, stencil_masks, full_stencil_size);
            if (err != cudaSuccess) {
                std::stringstream temp;
                temp << "Copy stencil to constant memory returned error code " << cudaGetErrorString(err);
                throw std::runtime_error(temp.str());
            }
        }
        void copy_constants_to_p2p_constant_memory(const double *constants, const size_t constants_size) {
            cudaError_t err = cudaMemcpyToSymbol(device_four_constants, constants, constants_size);
            if (err != cudaSuccess) {
                std::stringstream temp;
                temp << "Copy four-constants to constant memory returned error code " << cudaGetErrorString(err);
                throw std::runtime_error(temp.str());
            }
        }


        __device__ constexpr size_t component_length = ENTRIES + SOA_PADDING;
        __device__ constexpr size_t component_length_unpadded = INNER_CELLS + SOA_PADDING;

        __global__ void
        __launch_bounds__(64, 8)
        cuda_p2p_interactions_kernel(
            const double (&local_monopoles)[NUMBER_LOCAL_MONOPOLE_VALUES],
            double (&potential_expansions)[NUMBER_POT_EXPANSIONS_SMALL],
            const double theta, const double dx) {
            __shared__ double monopole_cache[8 * 18];
            __shared__ multiindex<> coarse_index_cache[8 * 18];
            // get local id
            int local_id = threadIdx.y * 8 + threadIdx.z;

            // use in case of debug prints
            // bool first_thread = (blockIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0);
            // Set cell indices
            const octotiger::fmm::multiindex<> cell_index((threadIdx.x + blockIdx.x * 1) + INNER_CELLS_PADDING_DEPTH,
                threadIdx.y + INNER_CELLS_PADDING_DEPTH, threadIdx.z + INNER_CELLS_PADDING_DEPTH);
            octotiger::fmm::multiindex<> cell_index_coarse(cell_index);
            cell_index_coarse.transform_coarse();
            const size_t cell_flat_index = octotiger::fmm::to_flat_index_padded(cell_index);
            octotiger::fmm::multiindex<> cell_index_unpadded((threadIdx.x + blockIdx.x * 1), threadIdx.y, threadIdx.z);
            const size_t cell_flat_index_unpadded =
                octotiger::fmm::to_inner_flat_index_not_padded(cell_index_unpadded);
            const int cache_index_base = cell_index_unpadded.y * 18 +
                            cell_index.z - 3;

            // Required for mask
            const double theta_rec_squared = sqr(1.0 / theta);
            const double d_components[2] = {1.0 / dx, -1.0 / dx};
            double tmpstore[4] = {0.0, 0.0, 0.0, 0.0};

            for (int stencil_x = STENCIL_MIN; stencil_x <= STENCIL_MAX; stencil_x++) {
                int x = stencil_x - STENCIL_MIN;
                __syncthreads();
                if (local_id < 18) {
                    for (int i = 0; i < 8; i++) {
                        const multiindex<> partner_index(INNER_CELLS_PADDING_DEPTH + blockIdx.x + stencil_x,
                                                            INNER_CELLS_PADDING_DEPTH + STENCIL_MIN + i,
                                                            3 + local_id);
                        const size_t partner_flat_index = to_flat_index_padded(partner_index);
                        multiindex<> partner_index_coarse(partner_index);
                        partner_index_coarse.transform_coarse();
                        coarse_index_cache[18*i + local_id] = partner_index_coarse;
                        monopole_cache[18*i + local_id] = local_monopoles[partner_flat_index];
                    }
                }
                __syncthreads();
                for (int stencil_y = STENCIL_MIN; stencil_y <= STENCIL_MAX; stencil_y++) {
                    int y = stencil_y - STENCIL_MIN;
                    for (int stencil_z = STENCIL_MIN; stencil_z <= STENCIL_MAX; stencil_z++) {
                        const size_t index = x * STENCIL_INX * STENCIL_INX + y * STENCIL_INX + (stencil_z - STENCIL_MIN);
                        if (!device_stencil_masks[index]) {
                            continue;
                        }

                        int cache_index = cache_index_base + stencil_z;

                        // Create mask
                        const double theta_c_rec_squared = static_cast<double>(
                            distance_squared_reciprocal(cell_index_coarse, coarse_index_cache[cache_index]));
                        const bool mask_b = theta_rec_squared > theta_c_rec_squared;
                        double mask = mask_b ? 1.0 : 0.0;


                        double monopole = monopole_cache[cache_index] * mask * d_components[0];
                        const double four[4] = {device_four_constants[index * 4 + 0],
                                                device_four_constants[index * 4 + 1],
                                                device_four_constants[index * 4 + 2],
                                                device_four_constants[index * 4 + 3]};
                        tmpstore[0] = tmpstore[0] + four[0] * monopole;
                        tmpstore[1] = tmpstore[1] + four[1] * monopole * d_components[1];
                        tmpstore[2] = tmpstore[2] + four[2] * monopole * d_components[1];
                        tmpstore[3] = tmpstore[3] + four[3] * monopole * d_components[1];
                        // compute_monopole_interaction<double>(monopole, tmpstore, four, d_components);
                    }
                    if (stencil_y < STENCIL_MAX && local_id < 18) {
                        // move stencil
                        __syncthreads();
                        for (int i = 0; i < 7; i++) {
                            coarse_index_cache[18*i + local_id] = coarse_index_cache[18*(i + 1) + local_id];
                            monopole_cache[18*i + local_id] = monopole_cache[18*(i + 1) + local_id];
                            __syncthreads();
                        }
                        // Load new row
                        const multiindex<> partner_index(INNER_CELLS_PADDING_DEPTH + blockIdx.x + stencil_x,
                                                         INNER_CELLS_PADDING_DEPTH + (stencil_y + 1) + 7,
                                                            3 + local_id);
                        const size_t partner_flat_index = to_flat_index_padded(partner_index);
                        multiindex<> partner_index_coarse(partner_index);
                        partner_index_coarse.transform_coarse();
                        coarse_index_cache[18*7 + local_id] = partner_index_coarse;
                        monopole_cache[18*7 + local_id] = local_monopoles[partner_flat_index];
                        __syncthreads();
                    }
                }
            }

            // Store results in output arrays
            potential_expansions[cell_flat_index_unpadded] = tmpstore[0];
            potential_expansions[1 * component_length_unpadded +
                cell_flat_index_unpadded] = tmpstore[1];
            potential_expansions[2 * component_length_unpadded +
                cell_flat_index_unpadded] = tmpstore[2];
            potential_expansions[3 * component_length_unpadded +
                cell_flat_index_unpadded] = tmpstore[3];
        }
        __global__ void cuda_add_pot_blocks(
            double (&potential_expansions)[3 * NUMBER_POT_EXPANSIONS_SMALL]) {
            size_t id = threadIdx.x;
            potential_expansions[id] += potential_expansions[NUMBER_POT_EXPANSIONS_SMALL + id];
            potential_expansions[1 * component_length_unpadded + id] +=
                potential_expansions[1 * component_length_unpadded + NUMBER_POT_EXPANSIONS_SMALL +
                    id];
            potential_expansions[2 * component_length_unpadded + id] +=
                potential_expansions[2 * component_length_unpadded + NUMBER_POT_EXPANSIONS_SMALL +
                    id];
            potential_expansions[3 * component_length_unpadded + id] +=
                potential_expansions[3 * component_length_unpadded + NUMBER_POT_EXPANSIONS_SMALL +
                    id];
            potential_expansions[id] += potential_expansions[2 * NUMBER_POT_EXPANSIONS_SMALL + id];
            potential_expansions[1 * component_length_unpadded + id] +=
                potential_expansions[1 * component_length_unpadded + 2 * NUMBER_POT_EXPANSIONS_SMALL +
                    id];
            potential_expansions[2 * component_length_unpadded + id] +=
                potential_expansions[2 * component_length_unpadded + 2 * NUMBER_POT_EXPANSIONS_SMALL +
                    id];
            potential_expansions[3 * component_length_unpadded + id] +=
                potential_expansions[3 * component_length_unpadded + 2 * NUMBER_POT_EXPANSIONS_SMALL +
                    id];
        }
    }    // namespace monopole_interactions
}    // namespace fmm
}    // namespace octotiger
#endif
