
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_CUDA_ENABLED
#include "compute_kernel_templates.hpp"
#include "cuda_kernel_methods.hpp"
namespace octotiger {
namespace fmm {
    // This specialization is only required on cuda devices since T::value_type is not supported!
    template <>
    CUDA_CALLABLE_METHOD inline void multiindex<int32_t>::transform_coarse() {
        const int32_t patch_size = static_cast<int32_t>(INX);
        const int32_t subtract = static_cast<int32_t>(INX / 2);
        x = ((x + patch_size) >> 1) - subtract;
        y = ((y + patch_size) >> 1) - subtract;
        z = ((z + patch_size) >> 1) - subtract;
    }

    CUDA_CALLABLE_METHOD inline int32_t distance_squared_reciprocal(
        const multiindex<>& i, const multiindex<>& j) {
        return (sqr(i.x - j.x) + sqr(i.y - j.y) + sqr(i.z - j.z));
    }

    namespace multipole_interactions {
        constexpr size_t component_length = ENTRIES + SOA_PADDING;
        __global__ void cuda_multipole_interactions_kernel(
            double (&local_monopoles)[NUMBER_LOCAL_MONOPOLE_VALUES],
            double (&center_of_masses)[NUMBER_MASS_VALUES],
            double (&multipoles)[NUMBER_LOCAL_EXPANSION_VALUES],
            double (&potential_expansions)[NUMBER_POT_EXPANSIONS],
            double (&angular_corrections)[NUMBER_ANG_CORRECTIONS],
            octotiger::fmm::multiindex<> (&stencil)[STENCIL_SIZE],
            bool (&stencil_phases)[STENCIL_SIZE], double (&factor_half)[20],
            double (&factor_sixth)[20], double theta) {
            printf("yay %f", theta);

            // Set cell indices
            const octotiger::fmm::multiindex<> cell_index(threadIdx.x + INNER_CELLS_PADDING_DEPTH,
                threadIdx.y + INNER_CELLS_PADDING_DEPTH, threadIdx.z + INNER_CELLS_PADDING_DEPTH);
            octotiger::fmm::multiindex<> cell_index_coarse(cell_index);
            cell_index_coarse.transform_coarse();
            const size_t cell_flat_index = octotiger::fmm::to_flat_index_padded(cell_index);
            octotiger::fmm::multiindex<> cell_index_unpadded(threadIdx.x, threadIdx.y, threadIdx.z);
            const size_t cell_flat_index_unpadded =
                octotiger::fmm::to_inner_flat_index_not_padded(cell_index_unpadded);

            // Load multipoles for this cell
            double m_cell[20];
            m_cell[0] = multipoles[0 * component_length + cell_flat_index];
            m_cell[1] = multipoles[1 * component_length + cell_flat_index];
            m_cell[2] = multipoles[2 * component_length + cell_flat_index];
            m_cell[3] = multipoles[3 * component_length + cell_flat_index];
            m_cell[4] = multipoles[4 * component_length + cell_flat_index];
            m_cell[5] = multipoles[5 * component_length + cell_flat_index];
            m_cell[6] = multipoles[6 * component_length + cell_flat_index];
            m_cell[7] = multipoles[7 * component_length + cell_flat_index];
            m_cell[8] = multipoles[8 * component_length + cell_flat_index];
            m_cell[9] = multipoles[9 * component_length + cell_flat_index];
            m_cell[10] = multipoles[10 * component_length + cell_flat_index];
            m_cell[11] = multipoles[11 * component_length + cell_flat_index];
            m_cell[12] = multipoles[12 * component_length + cell_flat_index];
            m_cell[13] = multipoles[13 * component_length + cell_flat_index];
            m_cell[14] = multipoles[14 * component_length + cell_flat_index];
            m_cell[15] = multipoles[15 * component_length + cell_flat_index];
            m_cell[16] = multipoles[16 * component_length + cell_flat_index];
            m_cell[17] = multipoles[17 * component_length + cell_flat_index];
            m_cell[18] = multipoles[18 * component_length + cell_flat_index];
            m_cell[19] = multipoles[19 * component_length + cell_flat_index];
            double X[NDIM];
            X[0] = center_of_masses[cell_flat_index];
            X[1] = center_of_masses[1 * component_length + cell_flat_index];
            X[2] = center_of_masses[2 * component_length + cell_flat_index];

            // Create and set result arrays
            double tmpstore[20];
            double tmp_corrections[3];
            for (size_t i = 0; i < 20; ++i)
                tmpstore[i] = 0.0;
            for (size_t i = 0; i < 3; ++i)
                tmp_corrections[i] = 0.0;
            // Required for mask
            const double theta_rec_squared = sqr(1.0 / theta);

            // calculate interactions between this cell and each stencil element
            for (size_t stencil_index = 0; stencil_index < STENCIL_SIZE; stencil_index++) {
                // Get phase indicator (indicates whether multipole multipole interactions still
                // needs to be done)
                const double mask_phase_one = static_cast<double>(stencil_phases[stencil_index]);

                // Get interaction partner indices
                const multiindex<>& stencil_element = stencil[stencil_index];
                const multiindex<> partner_index(cell_index.x + stencil_element.x,
                    cell_index.y + stencil_element.y, cell_index.z + stencil_element.z);
                const size_t partner_flat_index = to_flat_index_padded(partner_index);
                multiindex<> partner_index_coarse(partner_index);
                partner_index_coarse.transform_coarse();

                // Create mask - TODO is this really necessay in the non-vectorized code..?
                const double theta_c_rec_squared = static_cast<double>(
                    distance_squared_reciprocal(cell_index_coarse, partner_index_coarse));
                const bool mask_b = theta_rec_squared > theta_c_rec_squared;
                double mask = mask_b ? 1.0 : 0.0;

                // Load data of interaction partner
                double Y[NDIM];
                Y[0] = center_of_masses[partner_flat_index];
                Y[1] = center_of_masses[1 * component_length + partner_flat_index];
                Y[2] = center_of_masses[2 * component_length + partner_flat_index];
                double m_partner[20];
                m_partner[0] = local_monopoles[partner_flat_index] * mask;
                mask = mask * mask_phase_one;    // do not load multipoles outside the inner stencil
                m_partner[0] += multipoles[partner_flat_index] * mask;
                m_partner[1] = multipoles[1 * component_length + partner_flat_index] * mask;
                m_partner[2] = multipoles[2 * component_length + partner_flat_index] * mask;
                m_partner[3] = multipoles[3 * component_length + partner_flat_index] * mask;
                m_partner[4] = multipoles[4 * component_length + partner_flat_index] * mask;
                m_partner[5] = multipoles[5 * component_length + partner_flat_index] * mask;
                m_partner[6] = multipoles[6 * component_length + partner_flat_index] * mask;
                m_partner[7] = multipoles[7 * component_length + partner_flat_index] * mask;
                m_partner[8] = multipoles[8 * component_length + partner_flat_index] * mask;
                m_partner[9] = multipoles[9 * component_length + partner_flat_index] * mask;
                m_partner[10] = multipoles[10 * component_length + partner_flat_index] * mask;
                m_partner[11] = multipoles[11 * component_length + partner_flat_index] * mask;
                m_partner[12] = multipoles[12 * component_length + partner_flat_index] * mask;
                m_partner[13] = multipoles[13 * component_length + partner_flat_index] * mask;
                m_partner[14] = multipoles[14 * component_length + partner_flat_index] * mask;
                m_partner[15] = multipoles[15 * component_length + partner_flat_index] * mask;
                m_partner[16] = multipoles[16 * component_length + partner_flat_index] * mask;
                m_partner[17] = multipoles[17 * component_length + partner_flat_index] * mask;
                m_partner[18] = multipoles[18 * component_length + partner_flat_index] * mask;
                m_partner[19] = multipoles[19 * component_length + partner_flat_index] * mask;

                // Do all of the numeric stuff
                compute_kernel_rho<double>(
                    X, Y, m_partner, tmpstore, tmp_corrections, m_cell, factor_half,
                    factor_sixth);
            }
        }
    }    // namespace multipole_interactions
}    // namespace fmm
}    // namespace octotiger
#endif
