
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_CUDA_ENABLED
#include "cuda_kernel_methods.hpp"
namespace octotiger {
namespace fmm {
    // This specialization is only required on cuda devices since T::value_type is not supported!
    template <>
    CUDA_CALLABLE_METHOD inline void multiindex<int32_t>::transform_coarse() {
        const int32_t patch_size = static_cast<int32_t>(INX);
        const int32_t subtract = static_cast<int32_t>(INX / 2);
        x = ((x + patch_size) >> 1) - subtract;
        y = ((y + patch_size) >> 1) - subtract;
        z = ((z + patch_size) >> 1) - subtract;
    }

    CUDA_CALLABLE_METHOD inline int32_t distance_squared_reciprocal(
        const multiindex<>& i, const multiindex<>& j) {
        return (sqr(i.x - j.x) + sqr(i.y - j.y) + sqr(i.z - j.z));
    }

    namespace multipole_interactions {
        constexpr size_t padded_entries_per_component = ENTRIES + SOA_PADDING;
        __global__ void cuda_multipole_interactions_kernel(double* local_monopoles,
            double* center_of_masses, double* multipoles, double* potential_expansions,
            double* angular_corrections, octotiger::fmm::multiindex<>* stencil,
            bool* stencil_phases, double* factor_half, double* factor_sixth, double theta) {
            printf("yay %f", theta);

            // Set cell indices
            const octotiger::fmm::multiindex<> cell_index(threadIdx.x + INNER_CELLS_PADDING_DEPTH,
                threadIdx.y + INNER_CELLS_PADDING_DEPTH, threadIdx.z + INNER_CELLS_PADDING_DEPTH);
            octotiger::fmm::multiindex<> cell_index_coarse(cell_index);
            cell_index_coarse.transform_coarse();
            const size_t cell_flat_index = octotiger::fmm::to_flat_index_padded(cell_index);
            octotiger::fmm::multiindex<> cell_index_unpadded(threadIdx.x, threadIdx.y, threadIdx.z);
            const size_t cell_flat_index_unpadded =
                octotiger::fmm::to_inner_flat_index_not_padded(cell_index_unpadded);

            // Create and set result arrays
            double tmpstore[20];
            double tmp_corrections[20];
            for (size_t i = 0; i < 20; ++i) {
                tmpstore[i] = 0.0;
                tmp_corrections[i] = 0.0;
            }
            // Required for mask
            const double theta_rec_squared = sqr(1.0 / theta);

            // calculate interactions between this cell and each stencil element
            for (size_t stencil_index = 0; stencil_index < 743; stencil_index++) {
                // Get phase indicator (indicates whether multipole multipole interactions still
                // needs to be done)
                const double mask_phase_one = static_cast<double>(stencil_phases[stencil_index]);

                // Get interaction partner indices
                const multiindex<>& stencil_element = stencil[stencil_index];
                const multiindex<> interaction_partner_index(cell_index.x + stencil_element.x,
                    cell_index.y + stencil_element.y, cell_index.z + stencil_element.z);
                const size_t interaction_partner_flat_index =
                    to_flat_index_padded(interaction_partner_index);
                multiindex<> interaction_partner_index_coarse(interaction_partner_index);
                interaction_partner_index_coarse.transform_coarse();

                // Create mask - TODO is this really necessay in the non-vectorized code..?
                const double theta_c_rec_squared = static_cast<double>(distance_squared_reciprocal(
                    cell_index_coarse, interaction_partner_index_coarse));
                const bool mask_b = theta_rec_squared > theta_c_rec_squared;
                double mask = mask_b ? 1.0 : 0.0;

                double Y[NDIM];
                Y[0] = center_of_masses[interaction_partner_flat_index];
                Y[1] = center_of_masses[1 * padded_entries_per_component +
                    interaction_partner_flat_index];
                Y[2] = center_of_masses[2 * padded_entries_per_component +
                    interaction_partner_flat_index];

                double m_partner[20];
                m_partner[0] = local_monopoles[interaction_partner_flat_index] * mask;
                mask = mask * mask_phase_one;    // do not load multipoles outside the inner stencil
                m_partner[0] += multipoles[interaction_partner_flat_index] * mask;
                m_partner[1] =
                    multipoles[1 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[2] =
                    multipoles[2 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[3] =
                    multipoles[3 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[4] =
                    multipoles[4 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[5] =
                    multipoles[5 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[6] =
                    multipoles[6 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[7] =
                    multipoles[7 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[8] =
                    multipoles[8 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[9] =
                    multipoles[9 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[10] =
                    multipoles[10 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[11] =
                    multipoles[11 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[12] =
                    multipoles[12 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[13] =
                    multipoles[13 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[14] =
                    multipoles[14 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[15] =
                    multipoles[15 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[16] =
                    multipoles[16 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[17] =
                    multipoles[17 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[18] =
                    multipoles[18 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
                m_partner[19] =
                    multipoles[19 * padded_entries_per_component + interaction_partner_flat_index] *
                    mask;
            }
        }
    }    // namespace multipole_interactions
}    // namespace fmm
}    // namespace octotiger
#endif
