
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_CUDA_ENABLED
#include "cuda_kernel_methods.hpp"
namespace octotiger {
namespace fmm {
    // This specialization is only required on cuda devices since T::value_type is not supported!
    template <>
    CUDA_CALLABLE_METHOD inline void multiindex<int32_t>::transform_coarse() {
        const int32_t patch_size = static_cast<int32_t>(INX);
        const int32_t subtract = static_cast<int32_t>(INX / 2);
        x = ((x + patch_size) >> 1) - subtract;
        y = ((y + patch_size) >> 1) - subtract;
        z = ((z + patch_size) >> 1) - subtract;
    }

    CUDA_CALLABLE_METHOD inline int32_t distance_squared_reciprocal(
        const multiindex<>& i, const multiindex<>& j) {
        return (sqr(i.x - j.x) + sqr(i.y - j.y) + sqr(i.z - j.z));
    }

    namespace multipole_interactions {
        __global__ void cuda_multipole_interactions_kernel(double* center_of_masses,
            double* multipoles, double* potential_expansions, double* angular_corrections,
            octotiger::fmm::multiindex<>* stencil, bool* stencil_phases, double* factor_half,
            double* factor_sixth, double theta) {
            printf("yay %f", theta);

            // Set cell indices
            const octotiger::fmm::multiindex<> cell_index(threadIdx.x + INNER_CELLS_PADDING_DEPTH,
                threadIdx.y + INNER_CELLS_PADDING_DEPTH, threadIdx.z + INNER_CELLS_PADDING_DEPTH);
            octotiger::fmm::multiindex<> cell_index_coarse(cell_index);
            cell_index_coarse.transform_coarse();
            const size_t cell_flat_index = octotiger::fmm::to_flat_index_padded(cell_index);
            octotiger::fmm::multiindex<> cell_index_unpadded(threadIdx.x, threadIdx.y, threadIdx.z);
            const size_t cell_flat_index_unpadded =
                octotiger::fmm::to_inner_flat_index_not_padded(cell_index_unpadded);

            // Create and set result arrays
            double tmpstore[20];
            double tmp_corrections[20];
            for (size_t i = 0; i < 20; ++i) {
                tmpstore[i] = 0.0;
                tmp_corrections[i] = 0.0;
            }
            // Required for mask
            const double theta_rec_squared = sqr(1.0 / theta);

            // calculate interactions between this cell and each stencil element
            for (size_t stencil_index = 0; stencil_index < 743; stencil_index++) {
                // Get phase indicator (indicates whether multipole multipole interactions still
                // needs to be done)
                const bool phase_one = stencil_phases[stencil_index];

                // Get interaction partner indices
                const multiindex<>& stencil_element = stencil[stencil_index];
                const multiindex<> interaction_partner_index(cell_index.x + stencil_element.x,
                    cell_index.y + stencil_element.y, cell_index.z + stencil_element.z);
                const size_t interaction_partner_flat_index =
                    to_flat_index_padded(interaction_partner_index);
                multiindex<> interaction_partner_index_coarse(interaction_partner_index);
                interaction_partner_index_coarse.transform_coarse();

                // Create mask - TODO is this really necessay in the non-vectorized code..?
                const double theta_c_rec_squared = static_cast<double>(distance_squared_reciprocal(
                    cell_index_coarse, interaction_partner_index_coarse));
                const bool mask_b = theta_rec_squared > theta_c_rec_squared;
                const double mask = mask_b ? 1.0 : 0.0;
            }
        }
    }    // namespace multipole_interactions
}    // namespace fmm
}    // namespace octotiger
#endif
