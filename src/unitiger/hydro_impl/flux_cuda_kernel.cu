
#include <hip/hip_runtime.h>
#ifdef OCTOTIGER_HAVE_CUDA

#include <buffer_manager.hpp>
#include <cuda_buffer_util.hpp>
#include "octotiger/options.hpp"
#include "octotiger/cuda_util/cuda_helper.hpp"
#include <cuda_runtime.h>
#include <stream_manager.hpp>

#include "octotiger/unitiger/hydro_impl/flux_kernel_interface.hpp"

#include <hpx/synchronization/once.hpp>

__device__ inline int flip_dim(const int d, const int flip_dim) {
		int dims[3];
		int k = d;
		for (int dim = 0; dim < 3; dim++) {
			dims[dim] = k % 3;
			k /= 3;
		}
		k = 0;
		dims[flip_dim] = 2 - dims[flip_dim];
		for (int dim = 0; dim < 3; dim++) {
			k *= 3;
			k += dims[2 - dim];
		}
		return k;
}

__device__ const int faces[3][9] = { { 12, 0, 3, 6, 9, 15, 18, 21, 24 }, { 10, 0, 1, 2, 9, 11,
			18, 19, 20 }, { 4, 0, 1, 2, 3, 5, 6, 7, 8 } };

__device__ const int xloc[27][3] = {
	/**/{ -1, -1, -1 }, { +0, -1, -1 }, { +1, -1, -1 },
	/**/{ -1, +0, -1 }, { +0, +0, -1 }, { 1, +0, -1 },
	/**/{ -1, +1, -1 }, { +0, +1, -1 }, { +1, +1, -1 },
	/**/{ -1, -1, +0 }, { +0, -1, +0 }, { +1, -1, +0 },
	/**/{ -1, +0, +0 }, { +0, +0, +0 }, { +1, +0, +0 },
	/**/{ -1, +1, +0 }, { +0, +1, +0 }, { +1, +1, +0 },
	/**/{ -1, -1, +1 }, { +0, -1, +1 }, { +1, -1, +1 },
	/**/{ -1, +0, +1 }, { +0, +0, +1 }, { +1, +0, +1 },
	/**/{ -1, +1, +1 }, { +0, +1, +1 }, { +1, +1, +1 } };

__device__ const double quad_weights[9] = { 16. / 36., 1. / 36., 4. / 36., 1. / 36., 4. / 36., 4.
			/ 36., 1. / 36., 4. / 36., 1. / 36. };

hpx::lcos::local::once_flag flag1;

__host__ void init_gpu_masks(bool *masks) {
  auto masks_boost = create_masks();
  cudaMemcpy(masks, masks_boost.data(), NDIM * 1000 * sizeof(bool), cudaMemcpyHostToDevice);
}

__host__ const bool* get_gpu_masks(void) {
    static bool *masks = recycler::recycle_allocator_cuda_device<bool>{}.allocate(NDIM * 1000);
    hpx::lcos::local::call_once(flag1, init_gpu_masks, masks);
    return masks;
}

__device__ const int offset = 0;
__device__ const int compressedH_DN[3] = {100, 10, 1};
__device__ const int face_offset = 27 * 1000;
__device__ const int dim_offset = 1000;

__global__ void
__launch_bounds__(128, 2)
 flux_cuda_kernel(const double * __restrict__ q_combined, const double * __restrict__ x_combined, double * __restrict__ f_combined,
    double * amax, int * amax_indices, int * amax_d, const bool * __restrict__ masks, const double omega, const double dx, const double A_, const double B_, const double fgamma, const double de_switch_1) {
  __shared__ double sm_amax[128];
  __shared__ int sm_d[128];
  __shared__ int sm_i[128];

  // 3 dim 1000 i workitems
  //const int index = threadIdx.x * 100 + threadIdx.y * 10 + threadIdx.z + 100;

  const int dim = blockIdx.z;
  const int index = (blockIdx.y * 2 + threadIdx.x) * 64 + threadIdx.y * 8 + threadIdx.z + 104;
  const int tid = threadIdx.x * 64 + threadIdx.y * 8 + threadIdx.z;   
  const double mask = masks[index + dim * dim_offset];
  //if(tid == 0)
  // printf("starting...");
  const int nf = 15;

  double local_f[15] = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
  double local_x[3] = {0.0, 0.0, 0.0};
  double local_vg[3] = {0.0, 0.0, 0.0};
  for (int f = 0; f < nf; f++) {
      f_combined[dim * 15 * 1000 + f * 1000 + index] = 0.0;
  }

  double current_amax = 0.0;
  int current_d = 0;
  if(mask != 0.0) {
    for (int fi = 0; fi < 9; fi++) {    // 9
      double this_ap = 0.0, this_am = 0.0;    // tmps
      const int d = faces[dim][fi];
      const int flipped_dim = flip_dim(d, dim);
      for (int dim = 0; dim < 3; dim++) {
          local_x[dim] = x_combined[dim * 1000 + index] + (0.5 * xloc[d][dim] * dx);
      }
      local_vg[0] = -omega * (x_combined[1000 + index] + 0.5 * xloc[d][1] * dx);
      local_vg[1] = +omega * (x_combined[index] + 0.5 * xloc[d][0] * dx);
      local_vg[2] = 0.0;
      inner_flux_loop2<double>(omega, nf, A_, B_, q_combined, local_f, local_x, local_vg,
        this_ap, this_am, dim, d, dx, fgamma, de_switch_1, dim_offset * d + index, dim_offset * flipped_dim - compressedH_DN[dim] + index, face_offset);
      this_ap *= mask;
      this_am *= mask;
      const double amax_tmp = max_wrapper(this_ap, (-this_am));
      if (amax_tmp > current_amax) {
        current_amax = amax_tmp;
        current_d = d;
      }
      for (int f = 1; f < nf; f++) {
        f_combined[dim * 15 * 1000 + f * 1000 + index] += quad_weights[fi] * local_f[f];
        if (f > 9) {
          f_combined[dim * 15 * 1000 + index] += quad_weights[fi] * local_f[f];
        }
      }
   }
 }

 // Find maximum:
 sm_amax[tid] = current_amax;
 sm_d[tid] = current_d;
 sm_i[tid] = index;
 __syncthreads();
 for (int tid_border = 64; tid_border >= 32; tid_border /= 2) {
   if(tid < tid_border) {
     if (sm_amax[tid + tid_border] > sm_amax[tid]) {
       sm_amax[tid] = sm_amax[tid + tid_border];
       sm_d[tid] = sm_d[tid + tid_border];
       sm_i[tid] = sm_i[tid + tid_border];
     }
   }
   __syncthreads();
 }
 // Max reduction within one warps
 for (int tid_border = 16; tid_border >= 1; tid_border /= 2) {
   if(tid < tid_border) {
     if (sm_amax[tid + tid_border] > sm_amax[tid]) {
       sm_amax[tid] = sm_amax[tid + tid_border];
       sm_d[tid] = sm_d[tid + tid_border];
       sm_i[tid] = sm_i[tid + tid_border];
     }
   }
 }

 if (tid == 0) {
   amax[dim * 7 + blockIdx.y] = sm_amax[0];
   amax_indices[dim * 7 + blockIdx.y] = sm_i[0];
   amax_d[dim * 7 + blockIdx.y] = sm_d[0];
 }
 return;
}

timestep_t launch_flux_cuda(stream_interface<hpx::cuda::experimental::cuda_executor, pool_strategy>& executor,
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_q, double* device_q,
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_f,
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> &combined_x, double* device_x,
    safe_real omega, const size_t nf_, double dx, size_t device_id) {
    timestep_t ts;
    const cell_geometry<3, 8> geo;


    /*size_t device_id =
      stream_pool::get_next_device_id<hpx::cuda::experimental::cuda_executor,
      pool_strategy>();*/
    //size_t device_id = 0;

    recycler::cuda_device_buffer<double> device_f(NDIM * 15 * 1000 + 32, device_id);
    const bool *masks = get_gpu_masks();

    recycler::cuda_device_buffer<double> device_amax(7 * NDIM);
    recycler::cuda_device_buffer<int> device_amax_indices(7 * NDIM);
    recycler::cuda_device_buffer<int> device_amax_d(7 * NDIM);
    double A_ = physics<NDIM>::A_;
    double B_ = physics<NDIM>::B_;
    double fgamma = physics<NDIM>::fgamma_;
    double de_switch_1 = physics<NDIM>::de_switch_1;

    dim3 const grid_spec(1, 7, 3);
    dim3 const threads_per_block(2, 8, 8);
    void* args[] = {&(device_q),
      &(device_x), &(device_f.device_side_buffer), &(device_amax.device_side_buffer),
      &(device_amax_indices.device_side_buffer), &(device_amax_d.device_side_buffer), &masks, &omega, &dx, &A_, &B_, &fgamma, &de_switch_1};
    executor.post(
    cudaLaunchKernel<decltype(flux_cuda_kernel)>,
    flux_cuda_kernel, grid_spec, threads_per_block, args, 0);

    // Move data to host
    std::vector<double, recycler::recycle_allocator_cuda_host<double>> amax(7 * NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_indices(7 * NDIM);
    std::vector<int, recycler::recycle_allocator_cuda_host<int>> amax_d(7 * NDIM);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax.data(),
               device_amax.device_side_buffer, 7 * NDIM * sizeof(double),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_indices.data(),
               device_amax_indices.device_side_buffer, 7 * NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    hpx::apply(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, amax_d.data(),
               device_amax_d.device_side_buffer, 7 * NDIM * sizeof(int),
               cudaMemcpyDeviceToHost);
    auto fut = hpx::async(static_cast<hpx::cuda::experimental::cuda_executor>(executor),
               cudaMemcpyAsync, combined_f.data(), device_f.device_side_buffer,
               (NDIM * 15 * 1000 + 32) * sizeof(double), cudaMemcpyDeviceToHost);
    fut.get();

    // Find Maximum
    size_t current_id = 0;
    for (size_t i = 1; i < NDIM * 7; i++) {
      if (amax[i] > amax[current_id]) { 
        current_id = i;
      }
    }
    const size_t current_dim = current_id / 7;
    const size_t current_max_index = amax_indices[current_id];
    const size_t current_d = amax_d[current_id];
    //std::cin.get();
    std::vector<double> URs(nf_), ULs(nf_);
    ts.a = amax[current_dim];
    ts.x = combined_x[current_max_index];
    ts.y = combined_x[current_max_index + 1000];
    ts.z = combined_x[current_max_index + 2000];
    const auto flipped_dim = geo.flip_dim(current_d, current_dim);
    constexpr int compressedH_DN[3] = {100, 10, 1};
    for (int f = 0; f < nf_; f++) {
        URs[f] = combined_q[current_max_index + f * face_offset + dim_offset * current_d];
        ULs[f] = combined_q[current_max_index - compressedH_DN[current_dim] + f * face_offset +
            dim_offset * flipped_dim];
    }
    ts.ul = std::move(ULs);
    ts.ur = std::move(URs);
    ts.dim = current_dim;
    return ts;
}


#endif
